#include <hip/hip_runtime.h>


#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost
#define DTD hipMemcpyDeviceToDevice

__global__ void d_MatMul(float *A, float *B, float *C, int n)
{
	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((i<n) && (j<n)) {
        float value = 0.0;
		for (int k = 0; k < n; ++k) {
			value += A[i*n+k]*B[k*n+j];
		}
		C[i*n+j] = value;
	}
}

extern "C" void MatMul(float *A, float *B, float *C, int *N)
{

  float *dA, *dB, *dC;
  int n = *N;
  size_t sz_nn = n*n*sizeof(float);

  dim3 TPB(32,32), BPG;
  unsigned int bpg_x = (int) ceil((float)n/TPB.x);
  unsigned int bpg_y = (int) ceil((float)n/TPB.y);
  BPG.x = bpg_x;
  BPG.y = bpg_y;
	
	hipMalloc(&dA, sz_nn);
	hipMalloc(&dB, sz_nn);
	hipMalloc(&dC, sz_nn);
	hipMemcpy(dA, A, sz_nn, HTD);
	hipMemcpy(dB, B, sz_nn, HTD);

	d_MatMul<<<BPG, TPB>>>(dA, dB, dC, n);
	hipMemcpy(C, dC, sz_nn, DTH);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}

