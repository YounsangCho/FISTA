#include<hip/hip_runtime.h>

#include<hipblas.h>

#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost

extern "C"
void MatMul_cuBLAS(float *A, float *B, float *C, int *N)
{
    float *dA, *dB, *dC;
    int n = *N;
    size_t sz_1 = sizeof(float);
    size_t sz_nn = n*n*sizeof(float);
    float One = 1.0, Zero = 0.0;
    float *d_One, *d_Zero;

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    hipMalloc(&d_One, sz_1);
    hipMalloc(&d_Zero, sz_1);
    hipMalloc(&dA, sz_nn);
    hipMalloc(&dB, sz_nn);
    hipMalloc(&dC, sz_nn);

    hipMemcpy(d_One, &One, sz_1, HTD);
    hipMemcpy(d_Zero, &Zero, sz_1, HTD);
    hipMemcpy(dA, A, sz_nn, HTD);
    hipMemcpy(dB, B, sz_nn, HTD);

    hipblasCreate(&handle);
    hipblasSetPointerMode(handle,HIPBLAS_POINTER_MODE_DEVICE);
    stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 
                       d_One, dA, n, dB, n, d_Zero, dC, n);
    hipblasDestroy(handle);
    
    hipMemcpy(C, dC, sz_nn, DTH);
    
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}