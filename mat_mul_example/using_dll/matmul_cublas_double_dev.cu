#include<hip/hip_runtime.h>

#include "hipblas.h"

#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost

extern "C"
void MatMul_cuBLAS(double *A, double *B, double *C, int *N)
{
    double *dA, *dB, *dC;
    int n = *N;
    size_t sz = n*n*sizeof(double);
    const double One = 1.0;

    hipblasStatus_t stat;
    hipblasHandle_t handle;

    hipMalloc(&dA, sz);
    hipMalloc(&dB, sz);
    hipMalloc(&dC, sz);

    hipMemcpy(dA, A, sz, HTD);
    hipMemcpy(dB, B, sz, HTD);
    hipMemcpy(dC, C, sz, HTD);

    hipblasCreate(&handle);
    
    stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, 
                       &One, dA, n, dB, n, &One, dC, n);
    hipblasDestroy(handle);
    
    hipMemcpy(C, dC, sz, DTH);
    
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}