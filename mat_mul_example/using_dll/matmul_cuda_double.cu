#include <hip/hip_runtime.h>


#define HTD hipMemcpyHostToDevice
#define DTH hipMemcpyDeviceToHost
#define DTD hipMemcpyDeviceToDevice

__global__ void d_MatMul(double *A, double *B, double *C, int n)
{
	
	int i = blockIdx.y * blockDim.y + threadIdx.y;
	int j = blockIdx.x * blockDim.x + threadIdx.x;

	if ((i<n) && (j<n)) {
        double value = 0.0;
		for (int k = 0; k < n; ++k) {
			value += A[i*n+k]*B[k*n+j];
		}
		C[i*n+j] = value;
	}
}

extern "C" void MatMul(double *A, double *B, double *C, int *N)
{

	double *dA, *dB, *dC;
	int n = *N;
  size_t sz_nn = n*n*sizeof(double);

	dim3 TPB(32,32), BPG;
  unsigned int bpg_x = (int) ceil((double)n/TPB.x);
  unsigned int bpg_y = (int) ceil((double)n/TPB.y);
  BPG.x = bpg_x;
  BPG.y = bpg_y;
	
	hipMalloc(&dA, sz_nn);
	hipMalloc(&dB, sz_nn);
	hipMalloc(&dC, sz_nn);
	hipMemcpy(dA, A, sz_nn, HTD);
	hipMemcpy(dB, B, sz_nn, HTD);

	d_MatMul<<<BPG, TPB>>>(dA, dB, dC, n);
	hipMemcpy(C, dC, sz_nn, DTH);

	hipFree(dA);
	hipFree(dB);
	hipFree(dC);
}

